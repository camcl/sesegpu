#include "hip/hip_runtime.h"
#include "data.h"
#include <vector>
#include <chrono>
#include <iostream>
#include <thrust/functional.h>
#include <thrust/for_each.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>

namespace chrono = std::chrono;

__managed__ uint8_t* cudadataptr;
__managed__ int cudaelems;

struct computej
{
	int i;

	int __device__ operator() (int j)
	{
	  auto summer = [] (int a, int b)
	    {
	      int diff = a - b;
	      return diff * diff;
	    };
	  mnisttype data = mnisttype(cudadataptr, cudaelems);
	  return thrust::inner_product(thrust::device, &data(i, 0, 0), &data(i, 0, 0) + side * side, &data(j, 0, 0), 0, thrust::plus<int>(), summer);

	}
};

void __global__ dowork(int elems, int& maxi, int& maxj, int& divergence)
{
	thrust::counting_iterator<int32_t> zeroit(0);
	computej computer;

	auto domaxj = [] (int i)
	{
	        computej computer;
		computer.i = i / cudaelems;
		return computer(i % cudaelems);
	};

	auto transformer = make_transform_iterator(thrust::make_counting_iterator(0), domaxj);
	int maxval = thrust::max_element(thrust::seq, transformer, transformer + cudaelems * cudaelems) - transformer;
	maxj = maxval % cudaelems;
	maxi = maxval / cudaelems;

	computer.i = maxi;
	divergence = computer(maxj);
}


int main()
{
  printf("Thrust (max).\n");
  int elems = data.extent(0);
  cudaelems = elems;
  int* output;
  hipMallocManaged(&output, sizeof(int) * 3);
  
  int& maxi = output[0];
  int& maxj = output[1];
  int& divergence = output[2];


  hipMallocManaged(&cudadataptr, sizeof(uint8_t) * elems * side * side);
  printf("Procesing %d * %d elements\n", elems, elems);

  auto computestart = chrono::steady_clock::now();

  mnisttype localdata = mnisttype(cudadataptr, elems);
  hipError_t e;

  hipMemcpy(cudadataptr, data.data(), sizeof(uint8_t) * elems * side * side, hipMemcpyHostToDevice);
  hipMemAdvise(cudadataptr, sizeof(uint8_t) * elems * side * side, hipMemAdviseSetReadMostly, 0);
  dowork<<<1,1>>>(elems, maxi, maxj, divergence);

  e = hipDeviceSynchronize();
  if (e)
  {
	printf("Cuda error %d reported: %s\n", e, hipGetErrorString(e));
  }
  auto firstend = chrono::steady_clock::now();
  
  std::cout << "First pass, in microseconds : " << chrono::duration_cast<chrono::microseconds>(firstend-computestart).count() << std::endl;
  printf("Maximum divergence at %d against %d with value %d\n", maxi, maxj, divergence);
}
